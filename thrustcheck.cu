#include <iostream>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>


void is_a_thrust_bug() {
  std::cout << "START is_a_thrust_bug" << std::endl;

  std::cout << "CUDA LAST ERROR (0): "<<hipGetLastError()<<std::endl;

  const int N = 5105039;
  thrust::device_vector<int> p_srcs(N, 0);
  thrust::device_vector<int> p_dsts(N, 0);
  thrust::device_vector<float> p_weights(N, 0.0);

  std::cout << "CUDA LAST ERROR (1): "<<hipGetLastError()<<std::endl;

  std::cout << " -- stable_sort_by_key start" << std::endl;
  thrust::stable_sort_by_key(thrust::device,
                             p_dsts.begin(), p_dsts.end(),
                             thrust::make_zip_iterator(thrust::make_tuple(p_srcs.begin(), p_weights.begin())));
  std::cout << " -- stable_sort_by_key done" << std::endl;
  std::cout << "CUDA LAST ERROR (2): "<<hipGetLastError()<<std::endl;

  std::cout << "DONE is_a_thrust_bug" << std::endl;
}

#include <gunrock/gunrock.h>
void hits() {
   int x[2] = {0};
   int y[2] = {0};
   float a[2] = {0.0};
   float b[2] = {0.0};

   ::hits(0, 0, x, y, 0, a, b);
}
